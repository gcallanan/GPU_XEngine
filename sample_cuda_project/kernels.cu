
#include <hip/hip_runtime.h>
// Kernel function to add the elements of two arrays
__global__ void add_kernel(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}


void add(int N, float *x, float *y){
    add_kernel<<<1, 1>>>(N, x, y);
}